
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU under src!\n");
}

int main() {
    cuda_hello<<<1,1>>>(); 
    hipDeviceSynchronize();
    return 0;
}

