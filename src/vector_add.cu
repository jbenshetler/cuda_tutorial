
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>

constexpr int N = 1'000'000;

void vector_add_cpu(float* out, float* a, float* b, int n) {
    for (int i =0; i<n; i++) {
        out[i] = a[i] + b[i];
    }
}

__global__ void vector_add(float* out, float* a, float* b, int n) {
    for (int i =0; i<n; i++) {
        out[i] = a[i] + b[i];
    }
}


struct HostDevice {
    using T = float;
    using value_type = T;

    HostDevice(int const N)
    :
    N_{N}
    {
        host_ = reinterpret_cast<T*>( malloc( bytes() ) );
        hipMalloc(&dev_, bytes() );
    }

    ~HostDevice() {
        free(host_);
        host_ = nullptr;
        hipFree(dev_);
        dev_ = nullptr;
    }

    void copyHostToDevice() {
        hipMemcpy(dev_, host_, bytes(), hipMemcpyHostToDevice );
    }

    void copyDeviceToHost() {
        hipMemcpy(host_, dev_, bytes(), hipMemcpyDeviceToHost );
    }

    size_t bytes() const { return N_ * sizeof(value_type); }

    void fill(value_type const& value) {
        std::fill(host_, host_ + N, value);
    }

    int N_{0};
    T* host_{nullptr};
    T* dev_{nullptr};
};



int main(int argc, char* argv[]) {
    using namespace std;
    //vector<float> a(N, 1.0f);
    vector<float> b(N, 2.0f);
    vector<float> out(N, 0.0f);
    float* a = reinterpret_cast<float*>( malloc( sizeof(float) * N ) );

    float* dev_a;
    float* dev_b;
    float* dev_out;

    hipMalloc(&dev_a, sizeof(float) * N);
    hipMemcpy(dev_a, &a[0], sizeof(float)*N, ::hipMemcpyHostToDevice);

    hipMalloc(&dev_b, sizeof(float) * N);
    hipMemcpy(dev_b, &a[0], sizeof(float)*N, ::hipMemcpyHostToDevice);

    hipMalloc(&dev_out, sizeof(float) * N);
    hipMemcpy(dev_out, &a[0], sizeof(float)*N, ::hipMemcpyHostToDevice);


    vector_add<<<1,1>>>(&dev_out[0], &dev_a[0], &dev_b[0], N);

    hipMemcpy(&out[0], dev_out, sizeof(float)*N, ::hipMemcpyDeviceToHost);


    return 0;
}


