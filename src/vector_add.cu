#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>

#include "HostDevice.hpp"

constexpr int N = 1'000'000;

void vector_add_cpu(float* out, float* a, float* b, int n) {
    for (int i =0; i<n; i++) {
        out[i] = a[i] + b[i];
    }
}

__global__ void vector_add(float* out, float* a, float* b, int n) {
    for (int i =0; i<n; i++) {
        out[i] = a[i] + b[i];
    }
}



int main(int argc, char* argv[]) {
    using namespace std;

    HostDevice a(N);
    HostDevice b(N);
    HostDevice out(N);

    a.fill(1.0f);
    b.fill(2.0f);

    a.copyHostToDevice();
    b.copyHostToDevice();


    vector_add<<<1,1>>>(a.dev(), b.dev(), out.dev(), N);

    out.copyDeviceToHost();

    return 0;
}


