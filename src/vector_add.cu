
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>

constexpr int N = 1'000'000;

void vector_add_cpu(float* out, float* a, float* b, int n) {
    for (int i =0; i<n; i++) {
        out[i] = a[i] + b[i];
    }
}

__global__ void vector_add(float* out, float* a, float* b, int n) {
    for (int i =0; i<n; i++) {
        out[i] = a[i] + b[i];
    }
}


struct HostDevice {
    using T = float;
    using value_type = T;

    HostDevice(int const N)
    :
    N_{N}
    {
        host_ = reinterpret_cast<T*>( malloc( bytes() ) );
        hipMalloc(&dev_, bytes() );
    }

    ~HostDevice() {
        free(host_);
        host_ = nullptr;
        hipFree(dev_);
        dev_ = nullptr;
    }

    void copyHostToDevice() {
        hipMemcpy(dev_, host_, bytes(), hipMemcpyHostToDevice );
    }

    void copyDeviceToHost() {
        hipMemcpy(host_, dev_, bytes(), hipMemcpyDeviceToHost );
    }

    size_t bytes() const { return N_ * sizeof(value_type); }

    void fill(value_type const& value) {
        std::fill(host_, host_ + N, value);
    }

    int N_{0};
    T* host_{nullptr};
    T* dev_{nullptr};
};



int main(int argc, char* argv[]) {
    using namespace std;

    HostDevice a(N);
    HostDevice b(N);
    HostDevice out(N);

    a.fill(1.0f);
    b.fill(2.0f);

    a.copyHostToDevice();
    b.copyHostToDevice();


    vector_add<<<1,1>>>(a.dev_, b.dev_, out.dev_, N);

    out.copyDeviceToHost();

    return 0;
}


